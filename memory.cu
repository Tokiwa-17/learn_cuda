#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *ptr) {
    *ptr = 47;
}

int main() {
    int *ptr;
    checkCudaErrors(hipMalloc(&ptr, sizeof(int)));
    kernel<<<1, 1>>>(ptr);
    int ret;
    checkCudaErrors(hipMemcpy(&ret, ptr, sizeof(int), hipMemcpyDeviceToHost));
    printf("%d\n", ret);
    // checkCudaErrors(hipDeviceSynchronize());
    hipFree(ptr);
    return 0;
}