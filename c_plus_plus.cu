#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include <type_traits>

template <class T>
struct CudaAllocator {
    using value_type = T;
    T *allocate(size_t size) {
        T *ptr = nullptr;
        checkCudaErrors(hipMallocManaged(&ptr, size * sizeof(T)));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0) {
        checkCudaErrors(hipFree(ptr));
    }

    // template <class ...Args>
    // void construct(T *p, Args &&...args) {
    //     if constexpr (!(sizeof...(Args) == 0 && std::is_pod_v<T>))
    //         ::new((void *)p) T(std::forward<Args>(args)...);
    // }
};

template <int N, class T>
__global__ void kernel(T *arr) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        arr[i] = i;
    }
}

template <class T>
bool test(std::vector<T, CudaAllocator<T>> &ptr, int n) {
    bool flag = true;
    for (int i = 0; i < n; i++) {
        if (ptr[i] != i) {
            flag = false;
            break;
        }
    }
    return flag;
}

int main() {
    const int n = 65536;
    std::vector<int, CudaAllocator<int>> arr(n);

    kernel<n><<<32, 128>>>(arr.data());

    checkCudaErrors(hipDeviceSynchronize());
    
    if (test(arr, n)) {
        printf("cong.\n");
    } else {
        printf("failed.\n");
    }
    return 0;
}