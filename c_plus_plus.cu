#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>

template <class T>
struct CudaAllocator {
    using value_type = T;
    T *allocate(size_t size) {
        T *ptr = nullptr;
        checkCudaErrors(hipMallocManaged(&ptr, size * sizeof(T)));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0) {
        checkCudaErrors(hipFree(ptr));
    }
};

__global__ void kernel(int *arr, int n) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        arr[i] = i;
    }
}

bool test(std::vector<int, CudaAllocator<int>> &ptr, int n) {
    bool flag = true;
    for (int i = 0; i < n; i++) {
        if (ptr[i] != i) {
            flag = false;
            break;
        }
    }
    return flag;
}

int main() {
    int n = 65536;
    std::vector<int, CudaAllocator<int>> arr(n);

    kernel<<<32, 128>>>(arr.data(), n);

    checkCudaErrors(hipDeviceSynchronize());
    
    if (test(arr, n)) {
        printf("cong.\n");
    } else {
        printf("failed.\n");
    }
    return 0;
}