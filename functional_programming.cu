#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include <type_traits>

template <class Func>
__global__ void parallel_for(int n, Func func) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        func(i);
    }
}

struct MyFunctor {
    __device__ void operator() (int i) const {
        printf("number: %d\n", i);
    }
};

int main() {
    int n = 65536;
    parallel_for<<<32, 128>>> (n, MyFunctor{});
    checkCudaErrors(hipDeviceSynchronize());
    return 0;
}