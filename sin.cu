#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include <type_traits>
#include <iostream>
#include "ticktock.h"

#define EPS 1e-6

template <class Func>
__global__ void parallel_for(int n, Func func) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        func(i);
    }
}

struct MyFunctor {
    __device__ void operator() (int i) const {
        printf("number: %d\n", i);
    }
};

template <class T>
struct CudaAllocator {
    using value_type = T;
    T *allocate(size_t size) {
        T *ptr = nullptr;
        checkCudaErrors(hipMallocManaged(&ptr, size * sizeof(T)));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0) {
        checkCudaErrors(hipFree(ptr));
    }

    // template <class ...Args>
    // void construct(T *p, Args &&...args) {
    //     if constexpr (!(sizeof...(Args) == 0 && std::is_pod_v<T>))
    //         ::new((void *)p) T(std::forward<Args>(args)...);
    // }
};

template <class T, class Func>
bool test(std::vector<T, CudaAllocator<T>> &ptr, Func func, int n) {
    bool flag = true;
    for (int i = 0; i < n; i++) {
        if (ptr[i] - func(i) > EPS) {
            printf("i: %d, %f %f\n", i, ptr[i], func(i));
            flag = false;
            break;
        }
    }
    return flag;
}

int main() {
    int n = 65536;
    std::vector<float, CudaAllocator<float>> arr(n);
    TICK(cpu_sinf);
    parallel_for<<<32, 128>>> (n, [arr = arr.data()] __device__ (int i) {
        arr[i] = sinf(i);
    });
    TOCK(cpu_sinf);
    checkCudaErrors(hipDeviceSynchronize());
    if(test(arr, sinf, n)) printf("cong.\n");
    else printf("failed.\n");
    return 0;
}